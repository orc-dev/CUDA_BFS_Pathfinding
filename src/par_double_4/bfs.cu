#include "hip/hip_runtime.h"
/**
 * @file          bfs.cu
 * @brief         Implementation of a CUDA kernel for double-source BFS.
 *
 * @author        Xin Cai
 * @email         xcai72@wisc.edu
 * @date          Nov. 24, 2023
 *
 * @course        ME759: High Performance Computing for Engineering Application
 * @instructor    Professor Dan Negrut
 * @assignment    Final Project   
 */

#include "bfs.cuh"


/**
 * @brief   CUDA kernel to copy values from a temporary buffer to a main buffer.
 *
 * @param   omap  Pointer to the main buffer.
 */
__global__ void kernel_cpy(int* omap) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // copy from temp buffer to main buffer
    if (omap[idx] == DEF_VAL && omap[idx + NSQ] < DEF_VAL)
        omap[idx] = omap[idx + NSQ];
}


/**
 * @brief   CUDA kernel for breadth-first search (BFS) exploration on a graph.
 *
 * @param   emap  Pointer to the encoded map.
 * @param   omap  Pointer to the output map.
 */
__global__ void kernel_bfs(const std::uint8_t* emap, int* omap) {
    // variables initialization
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= NSQ) return;

    const std::uint8_t k = emap[idx];
    // thread terminal check: current cell is either 'an obstacle'
    // or 'has been processed as part of the target subpath'.
    if ((k & OPEN) || omap[idx] < 0)
        return;
    
    // static data for neighbor directions and offsets
    const int dir[4] = { TOP, BOT, LHS, RHS };
    const int off[4] = {  -N,   N,  -1,   1 }; 
    int nid;

    // kernel Terminal check: verify if a cell in the source subpath 
    // has a neighboring cell in the target subpath.
    if (omap[idx] < DEF_VAL) {
        // check if the termination condition is met
        for (int i = 0; i < 4; ++i) {
            nid = idx + off[i];
            if ((k & dir[i]) == 0 && omap[nid] < 0) {  // check top
                omap[SID] = idx;
                omap[TID] = ~nid;
                return;
            }
        }
        return;
    }

    for (int i = 0; i < 4; ++i) {
        nid = idx + off[i];
        if ((k & dir[i]) == 0 && omap[nid] != DEF_VAL) {    
            omap[idx + NSQ] = (omap[nid] < 0) ? ~nid : nid;
            return;
        }
    }
}


/**
 * @brief   Host function for breadth-first search (BFS) on a graph.
 *
 * @param   emap   Pointer to the encoded map.
 * @param   omap   Pointer to the output map.
 */
__host__ void bfs(const std::uint8_t* emap, int* omap) {
    // constants for kernel configurations
    const int thd_num = 128;
    const int blk_num = (NSQ + thd_num - 1) / thd_num;

    // double-buffer and double kernel to do block-level synchronization
    while (omap[SID] == DEF_VAL) {
        kernel_bfs<<<blk_num, thd_num>>>(emap, omap);
        hipDeviceSynchronize();

        if (omap[SID] != DEF_VAL) break;

        kernel_cpy<<<blk_num, thd_num>>>(omap);
        hipDeviceSynchronize();
    }
}
